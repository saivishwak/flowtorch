
#include <hip/hip_runtime.h>
#include <math.h>

// TODO : Support column major format
#define UNARY_OP(TYPENAME, FN_NAME, FUNC)                                                    \
  extern "C" __global__ void FN_NAME(const size_t numel, const TYPENAME *lhs, TYPENAME *out) \
  {                                                                                          \
    for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel;                  \
         i += blockDim.x * gridDim.x)                                                        \
    {                                                                                        \
      TYPENAME x = lhs[i];                                                                   \
      out[i] = FUNC;                                                                         \
    }                                                                                        \
  }

// Neg
UNARY_OP(float, uneg_f32, -1 * x);
UNARY_OP(double, uneg_f64, -1 * x);
UNARY_OP(int, uneg_i32, -1 * x);
UNARY_OP(long long, uneg_i64, -1 * x);

// Sqr
UNARY_OP(float, usqr_f32, x *x);
UNARY_OP(double, usqr_f64, x *x);
UNARY_OP(int, usqr_i32, x *x);
UNARY_OP(long long, usqr_i64, x *x);

// Sqrt
UNARY_OP(float, usqrt_f32, sqrt(x));
UNARY_OP(double, usqrt_f64, sqrt(x));

// Ceil
UNARY_OP(float, uceil_f32, ceil(x));
UNARY_OP(double, uceil_f64, ceil(x));

// Floor
UNARY_OP(float, ufloor_f32, floor(x));
UNARY_OP(double, ufloor_f64, floor(x));